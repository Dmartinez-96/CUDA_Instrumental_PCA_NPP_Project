#include "hip/hip_runtime.h"
#include <npp.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <filesystem>
#include <map>
#include <algorithm>

// Constants
const int SIGNAL_LENGTH = 1024; // Can be adjusted based on input signal
const int FFT_LENGTH = SIGNAL_LENGTH;

// Feature extraction structure to store all the features for machine learning.
struct Features {
    float magnitude[SIGNAL_LENGTH]; // FFT Magnitudes
    float phase[SIGNAL_LENGTH];     // FFT Phases
    float spectralCentroid;         // Spectral Centroid
    float spectralFlatness;         // Spectral Flatness
    float spectralBandwidth;        // Spectral Bandwidth
    float zcr;                      // Zero Crossing Rate
    float energy;                   // Signal Energy
    float temporal_mean;            // Temporal Mean
    float temporal_kurtosis;        // Temporal Variance
    float temporal_skewness;        // Temporal Skewness
    float temporal_variance;        // Temporal Variance
};

// Convert string to lowercase
std::string toLowerCase(const std::string& str) {
    std::string lowerStr = str;
    std::transform(lowerStr.begin(), lowerStr.end(), lowerStr.begin(), ::tolower);
    return lowerStr;
}

__global__ void scaleSignal(float* d_signal, float scale) {
    /*
    Documentation:
        Scales the input signal on the GPU by a specified factor using CUDA parallelization.

    Inputs:
        float* d_signal:
            - A pointer to the signal data stored on the device (GPU).
            - Each element in the signal is a floating-point number representing the signal's amplitude at a specific time point.

        float scale:
            - The scaling factor by which each element in the signal will be multiplied.
            - This value is applied uniformly across the entire signal.

    Outputs:
        void, no return:
            - The function modifies the input signal in place on the GPU, so there is no return value.
            - Each element in the `d_signal` array will be scaled by the specified factor.
    */

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < SIGNAL_LENGTH) {
        d_signal[idx] *= scale;
    }
}

// Calculate zero crossing rate with CUDA kernel.
__global__ void calculateZCR(float* d_signal, int length, float* d_zcr) {
    /*
    Documentation:
        Calculates the zero crossing rate for the signal.
    Inputs:
        float* d_signal:
            - A pointer to the signal data stored on the device (GPU).
            - Each element in the signal is a floating-point number representing the signal's amplitude at a specific time point.
        int length:
            - An integer representing the length of the input signal data array.
        float* d_zcr:
            - A pointer to the computed ZCR data stored on the device (GPU).
    Outputs:
        void, no return:
            - The function uses the input signal to calculate the ZCR and store the data in d_zcr.
    */
   int zero_crossings = 0;
    for (int i = 1; i < length; ++i) {
        if ((d_signal[i - 1] > 0 && d_signal[i] < 0) || (d_signal[i - 1] < 0 && d_signal[i] > 0)) {
            zero_crossings++;
        }
    }
    *d_zcr = static_cast<float>(zero_crossings) / length;
}


void fft_feature_extraction(float* h_signal, int length, Features* h_features) {
    /*
    Documentation:
        Extracts features from the input signal by performing a Fast Fourier Transform (FFT) and calculating the magnitude of the FFT result. This process is accelerated using Nvidia Performance Primitives (NPP) and CUDA.

    Inputs:
        float* h_signal:
            - A pointer to the signal data stored on the host (CPU).
            - The input signal is a time-domain signal, represented as an array of floating-point numbers.

        int length:
            - The length of the input signal array.
            - This value determines the number of elements to process in the FFT and feature extraction.

        float* h_features:
            - A pointer to the array where the extracted features will be stored on the host (CPU).
            - The array should be pre-allocated to have enough space to store the magnitude values resulting from the FFT.

    Outputs:
        void, no return:
            - The function performs FFT on the input signal and stores the extracted features (magnitude of the FFT) in the `h_features` array on the host.
            - No value is returned, but the `h_features` array is modified to contain the extracted features.
    */
   
   // Allocate device memory first.
   float *d_signal, *d_fft_result;
   hipMalloc((void**)&d_signal, length * sizeof(float));
   hipMalloc((void**)&d_fft_result, length * sizeof(Npp32fc)); 
   // This is a complex float for FFT

    // Copy signal from host to device
    hipMemcpy(d_signal, h_signal, length * sizeof(float), hipMemcpyHostToDevice);

    // Set up FFT configuration
    NppiFFTSpec_R_32f* pFFTSpec = nullptr;
    Npp8u* pBuffer = nullptr;
    int bufferSize = 0;
    nppiFFTGetSize_R_32f(NppiFFT_R_32f_SIZE, length, &bufferSize, nullptr);
    hipMalloc((void**)&pBuffer, bufferSize);
    nppiFFTInit_R_32f(&pFFTSpec, length, pBuffer);

    // Perform FFT
    nppStatus = nppiFFTFwd_RToCCS_32f(d_signal, 1, d_fft_result, pFFTSpec, pBuffer);
    if (nppStatus != NPP_SUCCESS) {
        std::cerr << "FFT failed!" << std::endl;
        hipFree(d_signal);
        hipFree(d_fft_result);
        return;
    }

    // Calculate magnitude and phase from FFT results
    nppiMagnitude_32fc(d_fft_result, 1, h_features->magnitude, 1, {length, 1});
    nppiPhase_32fc(d_fft_result, 1, h_features->phase, 1, {length, 1});

    // Calculate spectral centroid and bandwidth
    float spectral_centroid = 0.0f;
    float sum_magnitudes = 0.0f;
    for (int i = 0; i < length; ++i) {
        spectral_centroid += h_features->magnitude[i] * i;
        sum_magnitudes += h_features->magnitude[i];
    }
    h_features->spectral_centroid /= sum_magnitudes;

    float spectral_bandwidth = 0.0f;
    for (int i = 0; i < length; ++i) {
        spectral_bandwith += h_features->magnitude[i] * powf((i - h_features->spectral_centroid), 2);
    }
    h_features->spectral_bandwidth = sqrtf(spectral_bandwidth / sum_magnitudes);

    // Calculate spectral flatness
    float geom_mean = 1.0f;
    for (int i = 0; i < length; ++i) {
        geom_mean *= h_features->magnitude[i];
    }
    geom_mean = powf(geom_mean, 1.0f / length);
    
    float arithm_mean = sum_magnitudes / length;
    h_features->spectral_flatness = geom_mean / arith_mean;

    // Using CUDA kernel, calculate ZCR
    float* d_zcr;
    hipMalloc((void**)&d_zcr, sizeof(float));
    calculateZCR<<<1, 1>>>(d_signal, length, d_zcr);
    hipMemcpy(&h_features->zcr, d_zcr, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_zcr);

    // Calculate energy
    float energy = 0.0f;
    for (int i = 0; i < length; ++i) {
        energy += h_signal[i] * h_signal[i];
    }
    h_features->energy = sqrtf(energy / length);

    // Calculate temporal features
    float sum = 0.0f;
    for (int i = 0; i < length; ++i) {
        sum += h_signal[i];
    }
    h_features->temporal_mean = sum / length;

    float variance_sum = 0.0f;
    for (int i = 0; i < length; ++i) {
        variance_sum += powf(h_signal[i] - h_features->temporal_mean, 2.0f);
    }
    h_features->temporal_variance = variance_sum / length;

    float kurtosis_sum = 0.0f;
    for (int i = 0; i < length; ++i) {
        kurtosis_sum += powf(h_signal[i] - h_features->temporal_mean, 4.0f);
    }
    h_features->temporal_kurtosis = kurtosis_sum / (length * powf(h_features->temporal_variance, 2.0f)) - 3.0f;

    float skewness_sum = 0.0f;
    for (int i = 0; i < length; ++i) {
        skewness_sum += powf(h_signal[i] - h_features->temporal_mean, 3.0f);
    }
    h_features->temporal_skewness = skewness_sum / (length * powf(h_features->temporal_variance, 1.5f));

    // Cleanup!
    hipFree(d_signal);
    hipFree(d_fft_result);
    hipFree(pBuffer);
    nppiFFTFree_R_32f(pFFTSpec);
}

int main() {
    std::vector<std::string> wavFiles;
    std::map<std::string, std::string> fileToInstrumentMap;

    // Populate the wavFiles vector and fileToInstrumentMap based on filenames in data/WAV_files
    for (const auto& entry : std::filesystem::directory_iterator("data/WAV_files")) {
        if (entry.path().extension() == ".wav") {
            std::string filePath = entry.path().string();
            wavFiles.push_back(filePath);
            
            std::string lowerFilePath = toLowerCase(filePath);
            // Map each filename to an instrument label
            if (lowerFilePath.find("guitar") != std::string::npos || lowerFilePath.find("gtr") != std::string::npos) {
                fileToInstrumentMap[filePath] = "guitar";
            } else if (lowerFilePath.find("piano") != std::string::npos || lowerFilePath.find("pno") != std::string::npos) {
                fileToInstrumentMap[filePath] = "piano";
            } else if (lowerFilePath.find("violin") != std::string::npos || lowerFilePath.find("vln") != std::string::npos) {
                fileToInstrumentMap[filePath] = "violin";
            } else if (lowerFilePath.find("cello") != std::string::npos) {
                fileToInstrumentMap[filePath] = "cello";
            } else if (lowerFilePath.find("harpsichord") != std::string::npos || lowerFilePath.find("harpsi") != std::string::npos) {
                fileToInstrumentMap[filePath] = "harpsichord";
            } else if (lowerFilePath.find("gongs") != std::string::npos) {
                fileToInstrumentMap[filePath] = "gongs";
            } else if (lowerFilePath.find("bass") != std::string::npos) {
                fileToInstrumentMap[filePath] = "bass";
            } else if (lowerFilePath.find("marimba") != std::string::npos) {
                fileToInstrumentMap[filePath] = "marimba";
            } else if (lowerFilePath.find("oboe") != std::string::npos) {
                fileToInstrumentMap[filePath] = "oboe";
            } else if (lowerFilePath.find("shakuhachi") != std::string::npos) {
                fileToInstrumentMap[filePath] = "shakuhachi";
            } else if (lowerFilePath.find("sitar") != std::string::npos) {
                fileToInstrumentMap[filePath] = "sitar";
            } else if (lowerFilePath.find("flute") != std::string::npos) {
                fileToInstrumentMap[filePath] = "flute";
            } else if (lowerFilePath.find("sax") != std::string::npos) {
                fileToInstrumentMap[filePath] = "saxophone";
            } else if (lowerFilePath.find("trumpet") != std::string::npos) {
                fileToInstrumentMap[filePath] = "trumpet";
            } else if (lowerFilePath.find("viola") != std::string::npos) {
                fileToInstrumentMap[filePath] = "viola";
            }
        }
    }

    // Process each WAV file, extract features, and associate with instrument label
    std::vector<std::vector<float>> featuresMatrix;
    std::vector<std::string> labels;

    for (const auto& file : wavFiles) {
        float h_signal[SIGNAL_LENGTH];
        loadWavFileAquila(file.c_str(), h_signal, SIGNAL_LENGTH);

        Features h_features;
        fft_feature_extraction(h_signal, SIGNAL_LENGTH, &h_features);

        // Store extracted features
        std::vector<float> features_row = {
            h_features.spectralCentroid,
            h_features.spectralFlatness,
            h_features.spectralBandwidth,
            h_features.zcr,
            h_features.energy,
            h_features.temporal_mean,
            h_features.temporal_variance,
            h_features.temporal_skewness,
            h_features.temporal_kurtosis
        };
        featuresMatrix.push_back(features_row);

        // Store the corresponding instrument label
        labels.push_back(fileToInstrumentMap[file]);
    }

    // Optionally write the features and labels to a CSV file
    std::ofstream outFile("features_with_labels.csv");
    for (size_t i = 0; i < featuresMatrix.size(); ++i) {
        for (size_t j = 0; j < featuresMatrix[i].size(); ++j) {
            outFile << featuresMatrix[i][j];
            if (j < featuresMatrix[i].size() - 1) outFile << ",";
        }
        outFile << "," << labels[i] << "\n";
    }
    outFile.close();

    return 0;
}
