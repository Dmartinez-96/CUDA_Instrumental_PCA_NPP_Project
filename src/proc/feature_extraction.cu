#include "hip/hip_runtime.h"
#include <npp.h>
#include <nppi.h>
#include <npps.h>
#include <nppcore.h>
#include <nppdefs.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <filesystem>
#include <map>
#include <algorithm>
#include "../../include/feature_extraction.h"

// Constants
//const int SIGNAL_LENGTH = 1024; // Can be adjusted based on input signal

// Feature extraction structure to store all the features for machine learning.
struct Features {
    float magnitude[SIGNAL_LENGTH]; // FFT Magnitudes
    float phase[SIGNAL_LENGTH];     // FFT Phases
    float spectralCentroid;         // Spectral Centroid
    float spectralFlatness;         // Spectral Flatness
    float spectralBandwidth;        // Spectral Bandwidth
    float zcr;                      // Zero Crossing Rate
    float energy;                   // Signal Energy
    float temporalMean;            // Temporal Mean
    float temporalKurtosis;        // Temporal Variance
    float temporalSkewness;        // Temporal Skewness
    float temporalVariance;        // Temporal Variance
};

// Convert string to lowercase
std::string toLowerCase(const std::string& str) {
    std::string lowerStr = str;
    std::transform(lowerStr.begin(), lowerStr.end(), lowerStr.begin(), ::tolower);
    return lowerStr;
}

__global__ void scaleSignal(float* d_signal, float scale) {
    /*
    Documentation:
        Scales the input signal on the GPU by a specified factor using CUDA parallelization.

    Inputs:
        float* d_signal:
            - A pointer to the signal data stored on the device (GPU).
            - Each element in the signal is a floating-point number representing the signal's amplitude at a specific time point.

        float scale:
            - The scaling factor by which each element in the signal will be multiplied.
            - This value is applied uniformly across the entire signal.

    Outputs:
        void, no return:
            - The function modifies the input signal in place on the GPU, so there is no return value.
            - Each element in the `d_signal` array will be scaled by the specified factor.
    */

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < SIGNAL_LENGTH) {
        d_signal[idx] *= scale;
    }
}

// Calculate zero crossing rate with CUDA kernel.
__global__ void calculateZCR(float* d_signal, int length, float* d_zcr) {
    /*
    Documentation:
        Calculates the zero crossing rate for the signal.
    Inputs:
        float* d_signal:
            - A pointer to the signal data stored on the device (GPU).
            - Each element in the signal is a floating-point number representing the signal's amplitude at a specific time point.
        int length:
            - An integer representing the length of the input signal data array.
        float* d_zcr:
            - A pointer to the computed ZCR data stored on the device (GPU).
    Outputs:
        void, no return:
            - The function uses the input signal to calculate the ZCR and store the data in d_zcr.
    */
   int zero_crossings = 0;
    for (int i = 1; i < length; ++i) {
        if ((d_signal[i - 1] > 0 && d_signal[i] < 0) || (d_signal[i - 1] < 0 && d_signal[i] > 0)) {
            zero_crossings++;
        }
    }
    *d_zcr = static_cast<float>(zero_crossings) / length;
}


void fft_feature_extraction(float* h_signal, int length, Features* h_features) {
    /*
    Documentation:
        Extracts features from the input signal by performing a Fast Fourier Transform (FFT) and calculating the magnitude of the FFT result. This process is accelerated using Nvidia Performance Primitives (NPP) and CUDA.

    Inputs:
        float* h_signal:
            - A pointer to the signal data stored on the host (CPU).
            - The input signal is a time-domain signal, represented as an array of floating-point numbers.

        int length:
            - The length of the input signal array.
            - This value determines the number of elements to process in the FFT and feature extraction.

        float* h_features:
            - A pointer to the array where the extracted features will be stored on the host (CPU).
            - The array should be pre-allocated to have enough space to store the magnitude values resulting from the FFT.

    Outputs:
        void, no return:
            - The function performs FFT on the input signal and stores the extracted features (magnitude of the FFT) in the `h_features` array on the host.
            - No value is returned, but the `h_features` array is modified to contain the extracted features.
    */
   
    // Allocate device memory first.
    float *d_signal;
    hipfftComplex *d_fft_result;
    hipError_t err = hipMalloc((void**)&d_signal, length * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
    hipError_t err = hipMalloc((void**)&d_fft_result, length * sizeof(hipfftComplex)); 
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed: " << hipGetErrorString(err) << std::endl;
        hipFree(d_signal);
        return;
    }
    // This is a complex float for FFT

    // Create a 1D FFT plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, length, HIPFFT_R2C, 1);

    // Execute FFT plan
    hipfftExecR2C(plan, d_signal, d_fft_result);

    // Copy FFT result back to host
    hipfftComplex h_fft_result[length];
    hipMemcpy(h_fft_result, d_fft_result, length * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    // Calculate magnitude and phase from FFT results
    for (int i = 0; i < length; ++i) {
        h_features->magnitude[i] = sqrtf(h_fft_result[i].x * h_fft_result[i].x + h_fft_result[i].y * h_fft_result[i].y);
        h_features->phase[i] = atan2f(h_fft_result[i].y, h_fft_result[i].x);
    }

    // Calculate spectral centroid and bandwidth
    float spectral_centroid = 0.0f;
    float sum_magnitudes = 0.0f;
    for (int i = 0; i < length; ++i) {
        spectral_centroid += h_features->magnitude[i] * i;
        sum_magnitudes += h_features->magnitude[i];
    }
    h_features->spectralCentroid = spectral_centroid / sum_magnitudes;

    float spectral_bandwidth = 0.0f;
    for (int i = 0; i < length; ++i) {
        spectral_bandwidth += h_features->magnitude[i] * powf((i - h_features->spectralCentroid), 2);
    }
    h_features->spectralBandwidth = sqrtf(spectral_bandwidth / sum_magnitudes);

    // Calculate spectral flatness
    float geom_mean = 1.0f;
    for (int i = 0; i < length; ++i) {
        geom_mean *= h_features->magnitude[i];
    }
    geom_mean = powf(geom_mean, 1.0f / length);
    
    float arithm_mean = sum_magnitudes / length;
    h_features->spectralFlatness = geom_mean / arithm_mean;

    // Using CUDA kernel, calculate ZCR
    float* d_zcr;
    hipError_t err = hipMalloc((void**)&d_zcr, sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed: " << hipGetErrorString(err) << std::endl;
        hipfftDestroy(plan);
        hipFree(d_signal);
        hipFree(d_fft_result);
        return;
    }
    calculateZCR<<<1, 1>>>(d_signal, length, d_zcr);
    hipMemcpy(&h_features->zcr, d_zcr, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_zcr);

    // Calculate energy
    float energy = 0.0f;
    for (int i = 0; i < length; ++i) {
        energy += h_signal[i] * h_signal[i];
    }
    h_features->energy = sqrtf(energy / length);

    // Calculate temporal features
    float sum = 0.0f;
    for (int i = 0; i < length; ++i) {
        sum += h_signal[i];
    }
    h_features->temporalMean = sum / length;

    float variance_sum = 0.0f;
    for (int i = 0; i < length; ++i) {
        variance_sum += powf(h_signal[i] - h_features->temporalMean, 2.0f);
    }
    h_features->temporalVariance = variance_sum / length;

    float kurtosis_sum = 0.0f;
    for (int i = 0; i < length; ++i) {
        kurtosis_sum += powf(h_signal[i] - h_features->temporalMean, 4.0f);
    }
    h_features->temporalKurtosis = kurtosis_sum / (length * powf(h_features->temporalVariance, 2.0f)) - 3.0f;

    float skewness_sum = 0.0f;
    for (int i = 0; i < length; ++i) {
        skewness_sum += powf(h_signal[i] - h_features->temporalMean, 3.0f);
    }
    h_features->temporalSkewness = skewness_sum / (length * powf(h_features->temporalVariance, 1.5f));

    // Cleanup!
    hipfftDestroy(plan);
    hipFree(d_signal);
    hipFree(d_fft_result);
}