#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <npp.h>
#include <hipsolver.h>
#include "../../include/pca.h"

// Utility function to load the feature matrix from a CSV file
/*
    Description:
        This function loads a feature matrix from a CSV file into a 2D vector. 
        Each row of the CSV corresponds to a row in the feature matrix, and 
        each column corresponds to a feature.

    Inputs:
        const char* filename:
            - The path to the CSV file containing the feature matrix.
            - The CSV file should be formatted such that each row represents 
              a different data sample and each column represents a different feature.
              
        std::vector<std::vector<float>>& featuresMatrix:
            - A reference to a 2D vector where the loaded feature matrix will be stored.
            - The function will populate this vector with data from the CSV file.
    
    Outputs:
        void, no return:
            - The function loads the feature matrix into the provided vector, 
              modifying it directly. No return value is needed.
*/
void loadFeatureMatrix(const char* filename, std::vector<std::vector<float>>& featuresMatrix) {
    std::cout << "Loading feature matrix in pca.cu started." << std::endl;
    std::ifstream file(filename);
    std::string line;

    while (std::getline(file, line)) {
        std::vector<float> row;
        std::stringstream ss(line);
        std::string value;
        int colIndex = 0;
        bool has_nan = false;

        // Parse all columns except the last one (which is the label)
        while (std::getline(ss, value, ',')) {
            colIndex++;
            if (colIndex < 10) { // Assuming there are 9 numeric columns followed by the label
                try {
                    float num = std::stof(value);
                    if (std::isnan(num)) {
                        has_nan = true;
                        break;
                    }
                    row.push_back(num);
                } catch (const std::invalid_argument& e) {
                    std::cerr << "Invalid data encountered: " << value << std::endl;
                    has_nan = true;
                    break;
                }
            } else {
                // Skip the label
                break;
            }
        }

        if (!has_nan) {
            featuresMatrix.push_back(row);
        }
    }
    std::cout << "Loading feature matrix in pca.cu complete." << std::endl;
}

// Compute the covariance matrix using NPP and a custom CUDA kernel.

/*
    Description:
        This kernel manually computes the covariance matrix of a given feature matrix. 
        The feature matrix is first centered by subtracting the mean of each feature, 
        and then the covariance matrix is calculated.

    Inputs:
        float* d_centeredMatrix:
            - A pointer to the centered feature matrix on the device (GPU).
            - Each element is a floating-point value representing the deviation of a feature from its mean.

        float* d_covarianceMatrix:
            - A pointer to the covariance matrix on the device (GPU) where the computed covariance values will be stored.
            - This matrix will be populated with the covariance values for each pair of features.

        int num_samples:
            - The number of samples in the dataset.
            - This value determines how many data points are used in the covariance calculation.

        int num_features:
            - The number of features in the dataset.
            - This value determines the dimensions of the covariance matrix.
    
    Outputs:
        void, no return:
            - The function directly modifies the provided covariance matrix on the device with computed values.
*/
__global__ void computeCovarianceKernel(float* d_centeredMatrix, float* d_covarianceMatrix, int num_samples, int num_features) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_features && col < num_features) {
        float cov = 0.0f;
        for (int i = 0; i < num_samples; ++i) {
            cov += d_centeredMatrix[i * num_features + row] * d_centeredMatrix[i * num_features + col];
        }
        d_covarianceMatrix[row * num_features + col] = cov / (num_samples - 1);
        
        // Debug print to verify every element calculation
        printf("Thread [%d,%d] computed cov[%d][%d] = %f and stored at %d\n", 
               row, col, row, col, cov / (num_samples - 1), row * num_features + col);
    }
}


/*
    Description:
        This function computes the covariance matrix of a given feature matrix. 
        The feature matrix is first centered by subtracting the mean of each feature, 
        and then the covariance matrix is calculated.

    Inputs:
        const std::vector<std::vector<float>>& featuresMatrix:
            - A 2D vector where each row represents a data sample and each column 
              represents a feature.
            - The input matrix is expected to be centered before covariance 
              computation, but this function also handles centering.

        std::vector<std::vector<float>>& covarianceMatrix:
            - A reference to a 2D vector where the computed covariance matrix 
              will be stored.
            - The function will populate this vector with the covariance values 
              for each pair of features.
    
    Outputs:
        void, no return:
            - The function directly modifies the provided covariance matrix vector 
              with computed values. No return value is needed.
*/
void computeCovarianceMatrix(const std::vector<std::vector<float>>& featuresMatrix, std::vector<std::vector<float>>& covarianceMatrix) {
    std::cout << "Starting computeCovarianceMatrix in pca.cu." << std::endl;
    int num_samples = featuresMatrix.size();
    int num_features = NUM_FEATURES;

    // Center the feature matrix by subtracting the mean of each feature
    std::vector<float> means(num_features, 0.0f);
    for (int j = 0; j < num_features; ++j) {
        for (int i = 0; i < num_samples; ++i) {
            means[j] += featuresMatrix[i][j];
        }
        means[j] /= num_samples;
    }

    std::vector<std::vector<float>> centeredMatrix(num_samples, std::vector<float>(num_features));
    for (int j = 0; j < num_features; ++j) {
        for (int i = 0; i < num_samples; ++i) {
            centeredMatrix[i][j] = featuresMatrix[i][j] - means[j];
        }
    }

    std::cout << "Centered matrix calculated." << std::endl;

    // Convert centeredMatrix to a single array for NPP
    float* d_centeredMatrix = nullptr;
    hipError_t err = hipMalloc((void**)&d_centeredMatrix, num_samples * num_features * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for d_centeredMatrix: " << hipGetErrorString(err) << std::endl;
        return;
    }
    err = hipMemcpy(d_centeredMatrix, centeredMatrix[0].data(), num_samples * num_features * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy failed for d_centeredMatrix: " << hipGetErrorString(err) << std::endl;
        hipFree(d_centeredMatrix);
        return;
    }

    std::cout << "Centered matrix copied to device." << std::endl;

    float* d_covarianceMatrix = nullptr;
    err = hipMalloc((void**)&d_covarianceMatrix, num_features * num_features * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for d_covarianceMatrix: " << hipGetErrorString(err) << std::endl;
        hipFree(d_centeredMatrix);
        return;
    }

    std::cout << "Covariance matrix memory allocated on device." << std::endl;

    // Define grid and block sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((num_features + blockSize.x - 1) / blockSize.x, (num_features + blockSize.y - 1) / blockSize.y);
    std::cout << "Launching kernel with gridSize: " << gridSize.x << ", " << gridSize.y << std::endl;

    // Launch kernel to compute covariance matrix
    computeCovarianceKernel<<<gridSize, blockSize>>>(d_centeredMatrix, d_covarianceMatrix, num_samples, num_features);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed for computeCovarianceKernel: " << hipGetErrorString(err) << std::endl;
        hipFree(d_centeredMatrix);
        hipFree(d_covarianceMatrix);
        return;
    }

    std::cout << "Kernel launched successfully." << std::endl;

    // Synchronize to ensure kernel execution is complete
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA Device Synchronization failed after computeCovarianceKernel: " << hipGetErrorString(err) << std::endl;
        hipFree(d_centeredMatrix);
        hipFree(d_covarianceMatrix);
        return;
    }

    std::cout << "Kernel execution completed." << std::endl;

    // Copy the covariance matrix back to the host
    std::vector<float> h_covarianceMatrix(num_features * num_features);
    err = hipMemcpy(h_covarianceMatrix.data(), d_covarianceMatrix, num_features * num_features * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy failed for covarianceMatrix: " << hipGetErrorString(err) << std::endl;
        hipFree(d_centeredMatrix);
        hipFree(d_covarianceMatrix);
        return;
    }

    // Transfer data from the 1D vector to the 2D covarianceMatrix
    for (int i = 0; i < num_features; ++i) {
        for (int j = 0; j < num_features; ++j) {
            covarianceMatrix[i][j] = h_covarianceMatrix[i * num_features + j];
        }
    }

    // Debug: print the elements of the covariance matrix
    for (int i = 0; i < covarianceMatrix.size(); ++i) {
        for (int j = 0; j < covarianceMatrix[i].size(); ++j) {
            std::cout << "covarianceMatrix[" << i << "][" << j << "] = " << covarianceMatrix[i][j] << std::endl;
        }
    }

    // Free device memory
    hipFree(d_centeredMatrix);
    hipFree(d_covarianceMatrix);
    std::cout << "Finishing computeCovarianceMatrix in pca.cu." << std::endl;
}

// Perform eigenvalue decomposition using cuSolver
/*
    Description:
        This function performs eigenvalue decomposition on a covariance matrix.
        It calculates the eigenvalues and corresponding eigenvectors, which are 
        essential for Principal Component Analysis (PCA).

    Inputs:
        const std::vector<std::vector<float>>& covarianceMatrix:
            - A 2D vector representing the covariance matrix of the feature data.
            - The matrix is symmetric and square, with each dimension equal to 
              the number of features.

        std::vector<float>& eigenvalues:
            - A reference to a vector where the computed eigenvalues will be stored.
            - The function will populate this vector with the eigenvalues in 
              descending order of magnitude.

        std::vector<std::vector<float>>& eigenvectors:
            - A reference to a 2D vector where the computed eigenvectors will be stored.
            - Each column in this matrix corresponds to an eigenvector associated 
              with an eigenvalue in the `eigenvalues` vector.
    
    Outputs:
        void, no return:
            - The function modifies the `eigenvalues` and `eigenvectors` vectors 
              directly. No return value is needed.
*/
void performEigenDecomposition(const std::vector<std::vector<float>>& covarianceMatrix, std::vector<float>& eigenvalues, std::vector<std::vector<float>>& eigenvectors) {
    std::cout << "Starting performEigenDecomposition in pca.cu." << std::endl;
    int num_features = covarianceMatrix.size();

    // Allocate covariance matrix memory
    float* d_covarianceMatrix = nullptr;
    hipError_t err = hipMalloc((void**)&d_covarianceMatrix, num_features * num_features * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for d_covarianceMatrix: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // Flatten the host covariance matrix into a contiguous block of memory
    std::vector<float> h_covarianceMatrix(num_features * num_features);
    for (int i = 0; i < num_features; ++i) {
        std::copy(covarianceMatrix[i].begin(), covarianceMatrix[i].end(), h_covarianceMatrix.begin() + i * num_features);
    }

    // Copy the flattened covariance matrix to the device
    err = hipMemcpy(d_covarianceMatrix, h_covarianceMatrix.data(), num_features * num_features * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy failed for d_covarianceMatrix: " << hipGetErrorString(err) << std::endl;
        hipFree(d_covarianceMatrix);
        return;
    }

    // Allocate memory for eigenvalues and eigenvectors
    float* d_eigenvalues = nullptr;
    float* d_eigenvectors = nullptr;
    err = hipMalloc((void**)&d_eigenvalues, num_features * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for d_eigenvalues: " << hipGetErrorString(err) << std::endl;
        hipFree(d_covarianceMatrix);
        return;
    }
    err = hipMalloc((void**)&d_eigenvectors, num_features * num_features * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for d_eigenvectors: " << hipGetErrorString(err) << std::endl;
        hipFree(d_covarianceMatrix);
        hipFree(d_eigenvalues);
        return;
    }

    // Initialize solver
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t cusolver_status = hipsolverDnCreate(&cusolverH);
    if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
        std::cerr << "CUSOLVER initialization failed" << std::endl;
        hipFree(d_covarianceMatrix);
        hipFree(d_eigenvalues);
        hipFree(d_eigenvectors);
        return;
    }

    // Allocate space for solver workspace
    int work_size = 0;
    int* devInfo = NULL;
    err = hipMalloc((void**)&devInfo, sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for devInfo: " << hipGetErrorString(err) << std::endl;
        hipFree(d_covarianceMatrix);
        hipFree(d_eigenvalues);
        hipFree(d_eigenvectors);
        hipsolverDnDestroy(cusolverH);
        return;
    }

    // Allocate space for the workspace
    cusolver_status = hipsolverDnSsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, num_features, d_covarianceMatrix, num_features, d_eigenvalues, &work_size);
    if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
        std::cerr << "Failed to compute buffer size for eigenvalue decomposition" << std::endl;
        hipFree(d_covarianceMatrix);
        hipFree(d_eigenvalues);
        hipFree(d_eigenvectors);
        hipFree(devInfo);
        hipsolverDnDestroy(cusolverH);
        return;
    }
    std::cout << "Buffer size for eigenvalue decomposition: " << work_size << std::endl;

    float* work = nullptr;
    err = hipMalloc((void**)&work, work_size * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for work: " << hipGetErrorString(err) << std::endl;
        hipFree(d_covarianceMatrix);
        hipFree(d_eigenvalues);
        hipFree(d_eigenvectors);
        hipsolverDnDestroy(cusolverH);
        hipFree(devInfo);
        return;
    }

    // Perform the eigenvalue decomposition
    std::cout << "Starting hipsolverDnSsyevd for eigenvalue decomposition." << std::endl;
    cusolver_status = hipsolverDnSsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, num_features, d_covarianceMatrix, num_features, d_eigenvalues, work, work_size, devInfo);
    if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
        std::cerr << "Failed to perform eigenvalue decomposition" << std::endl;
        hipFree(d_covarianceMatrix);
        hipFree(d_eigenvalues);
        hipFree(d_eigenvectors);
        hipFree(work);
        hipFree(devInfo);
        hipsolverDnDestroy(cusolverH);
        return;
    }

    // Check devInfo value
    int devInfo_h = 0;
    hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    if (devInfo_h != 0) {
        std::cerr << "Eigenvalue decomposition failed, devInfo: " << devInfo_h << std::endl;
        hipFree(d_covarianceMatrix);
        hipFree(d_eigenvalues);
        hipFree(d_eigenvectors);
        hipFree(work);
        hipFree(devInfo);
        hipsolverDnDestroy(cusolverH);
        return;
    }

    std::cout << "Eigenvalue decomposition completed." << std::endl;

    // Copy the results back to the host
    err = hipMemcpy(eigenvalues.data(), d_eigenvalues, num_features * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy failed for d_eigenvalues: " << hipGetErrorString(err) << std::endl;
        hipFree(d_covarianceMatrix);
        hipFree(d_eigenvalues);
        hipFree(d_eigenvectors);
        hipFree(work);
        hipFree(devInfo);
        hipsolverDnDestroy(cusolverH);
        return;
    }

    std::vector<float> h_eigenvectors(num_features * num_features);
    err = hipMemcpy(h_eigenvectors.data(), d_eigenvectors, num_features * num_features * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy failed for d_eigenvectors: " << hipGetErrorString(err) << std::endl;
        hipFree(d_covarianceMatrix);
        hipFree(d_eigenvalues);
        hipFree(d_eigenvectors);
        hipFree(work);
        hipFree(devInfo);
        hipsolverDnDestroy(cusolverH);
        return;
    }

    // Transfer data from the 1D vector to the 2D eigenvectors matrix
    for (int i = 0; i < num_features; ++i) {
        for (int j = 0; j < num_features; ++j) {
            eigenvectors[i][j] = h_eigenvectors[i * num_features + j];
        }
    }

    // Debug: print the first few eigenvectors
    for (int i = 0; i < num_features; ++i) {
        for (int j = 0; j < num_features; ++j) {
            std::cout << "eigenvectors[" << i << "][" << j << "] = " << eigenvectors[i][j] << std::endl;
        }
    }

    
    hipFree(work);
    hipFree(devInfo);
    hipFree(d_eigenvectors);
    hipFree(d_eigenvalues);
    hipFree(d_covarianceMatrix);
    hipsolverDnDestroy(cusolverH);
    std::cout << "Finishing performEigenDecomposition in pca.cu." << std::endl;
}

// Project the data onto the principal components
/*
    Description:
        This function projects the original feature matrix onto the principal 
        components determined by the eigenvectors. The result is a new matrix 
        where each row represents the data in the reduced feature space.

    Inputs:
        const std::vector<std::vector<float>>& featuresMatrix:
            - A 2D vector where each row represents a data sample and each column 
              represents a feature.
            - This is the original data that will be projected onto the principal 
              components.

        const std::vector<std::vector<float>>& eigenvectors:
            - A 2D vector where each column represents an eigenvector corresponding 
              to one of the principal components.
            - The data will be projected onto these eigenvectors.

        std::vector<std::vector<float>>& pca_result:
            - A reference to a 2D vector where the projected data will be stored.
            - The function will populate this vector with the data represented 
              in the new, reduced feature space.
    
    Outputs:
        void, no return:
            - The function directly modifies the `pca_result` vector with the 
              projection of the data onto the principal components. No return value is needed.
*/
void projectOntoPrincipalComponents(const std::vector<std::vector<float>>& featuresMatrix, const std::vector<std::vector<float>>& eigenvectors, std::vector<std::vector<float>>& pca_result) {
    std::cout << "Starting projectOntoPrincipalComponents in pca.cu." << std::endl;
    int num_samples = featuresMatrix.size();
    int num_components = eigenvectors.size();

    for (int i = 0; i < num_samples; ++i) {
        for (int j = 0; j < num_components; ++j) {
            pca_result[i][j] = 0;
            for (int k = 0; k < num_components; ++k) {
                pca_result[i][j] += featuresMatrix[i][k] * eigenvectors[k][j];
            }
            std::cout << "pca_result[" << i << "][" << j << "] = " << pca_result[i][j] << std::endl;
        }
    }
    std::cout << "Finishing projectOntoPrincipalComponents in pca.cu." << std::endl;
}
